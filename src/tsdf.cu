#include "hip/hip_runtime.h"
/*
*
* Tariq Abuhashim
* t.abuhashim@gmail.com
* July, 2019
*
* Adapted from Andy Zeng, Princeton University, 2016
*
*/

//#include "tsdf.cuh"
#include "tsdf.hpp"

// CUDA kernel function to integrate a TSDF voxel volume given depth images
__global__
void GpuIntegrate(float * cam_K, float * cam2base, float * depth_im, int im_height, int im_width, 
				int voxel_grid_dim_x, int voxel_grid_dim_y, int voxel_grid_dim_z, 
				float voxel_grid_origin_x, float voxel_grid_origin_y, float voxel_grid_origin_z, 
				float voxel_size, float trunc_margin, float * voxel_grid_TSDF, float * voxel_grid_weight) 
{
	int pt_grid_z = blockIdx.x;
	int pt_grid_y = threadIdx.x;
	for (int pt_grid_x = 0; pt_grid_x < voxel_grid_dim_x; ++pt_grid_x) 
	{

		// Convert voxel center from grid coordinates to base frame camera coordinates
		float pt_base_x = voxel_grid_origin_x + pt_grid_x * voxel_size;
		float pt_base_y = voxel_grid_origin_y + pt_grid_y * voxel_size;
		float pt_base_z = voxel_grid_origin_z + pt_grid_z * voxel_size;

		// Convert from base frame camera coordinates to current frame camera coordinates
		float tmp_pt[3] = {0};
		tmp_pt[0] = pt_base_x - cam2base[0*4+3];
		tmp_pt[1] = pt_base_y - cam2base[1*4+3];
		tmp_pt[2] = pt_base_z - cam2base[2*4+3];
		float pt_cam_x = cam2base[0*4+0]*tmp_pt[0] + cam2base[1*4+0]*tmp_pt[1] + cam2base[2*4+0]*tmp_pt[2];
		float pt_cam_y = cam2base[0*4+1]*tmp_pt[0] + cam2base[1*4+1]*tmp_pt[1] + cam2base[2*4+1]*tmp_pt[2];
		float pt_cam_z = cam2base[0*4+2]*tmp_pt[0] + cam2base[1*4+2]*tmp_pt[1] + cam2base[2*4+2]*tmp_pt[2];
		if(pt_cam_z <= 0) continue;

		int pt_pix_x = roundf(cam_K[0*3+0] * (pt_cam_x / pt_cam_z) + cam_K[0*3+2]);
		int pt_pix_y = roundf(cam_K[1*3+1] * (pt_cam_y / pt_cam_z) + cam_K[1*3+2]);
		if(pt_pix_x < 0 || pt_pix_x >= im_width || pt_pix_y < 0 || pt_pix_y >= im_height) continue;

		float depth_val = depth_im[pt_pix_y * im_width + pt_pix_x];
		if(depth_val <= 0 || depth_val > 6) continue;

		float diff = depth_val - pt_cam_z;
		if(diff <= -trunc_margin) continue;

		// Integrate
		int volume_idx = pt_grid_z*voxel_grid_dim_y*voxel_grid_dim_x + pt_grid_y*voxel_grid_dim_x + pt_grid_x;
		float dist = fmin(1.0f, diff / trunc_margin);
		float weight_old = voxel_grid_weight[volume_idx];
		float weight_new = weight_old + 1.0f;
		voxel_grid_weight[volume_idx] = weight_new;
		voxel_grid_TSDF[volume_idx] = (voxel_grid_TSDF[volume_idx] * weight_old + dist) / weight_new;

	}
}

TSDF::TSDF(int h, int w, int MOid, std::vector<float> base2world_, std::vector<float> origin) : 
		im_height(h), im_width(w), mnId(MOid), base2world_vec(base2world_)
{
	// Location of voxel grid origin in base frame camera coordinates
	voxel_grid_origin_x = origin[0];
  	voxel_grid_origin_y = origin[1];
  	voxel_grid_origin_z = origin[2];
	// Read camera intrinsics
	std::copy(cam_K_vec.begin(), cam_K_vec.end(), cam_K);
	// Read base frame camera pose
	std::copy(base2world_vec.begin(), base2world_vec.end(), base2world);
	// Invert base frame camera pose to get world-to-base frame transform 
	invert_matrix(base2world, base2world_inv);

	// Initialize voxel grid
  	voxel_grid_TSDF = new float[voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z];
  	voxel_grid_weight = new float[voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z];
  	for (int i = 0; i < voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z; ++i)
    	voxel_grid_TSDF[i] = 1.0f;
  	memset(voxel_grid_weight, 0, sizeof(float) * voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z);

	// Load variables to GPU memory
	hipMalloc(&gpu_voxel_grid_TSDF, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float));
	hipMalloc(&gpu_voxel_grid_weight, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float));
	checkCUDA(__LINE__, hipGetLastError());
	hipMemcpy(gpu_voxel_grid_TSDF, voxel_grid_TSDF, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z *
								sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(gpu_voxel_grid_weight, voxel_grid_weight, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * 									sizeof(float), hipMemcpyHostToDevice);
	checkCUDA(__LINE__, hipGetLastError());
	hipMalloc(&gpu_cam_K, 3 * 3 * sizeof(float));
	hipMemcpy(gpu_cam_K, cam_K, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMalloc(&gpu_cam2base, 4 * 4 * sizeof(float));
	hipMalloc(&gpu_depth_im, im_height * im_width * sizeof(float));
	checkCUDA(__LINE__, hipGetLastError());
}

TSDF::~TSDF()
{
	// Load TSDF voxel grid from GPU to CPU memory
  	hipMemcpy(voxel_grid_TSDF, gpu_voxel_grid_TSDF, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * 
					sizeof(float), hipMemcpyDeviceToHost);
  	hipMemcpy(voxel_grid_weight, gpu_voxel_grid_weight, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * 
					sizeof(float), hipMemcpyDeviceToHost);
  	checkCUDA(__LINE__, hipGetLastError());

  	// Compute surface points from TSDF voxel grid and save to point cloud .ply file
  	//std::cout << "Saving surface point cloud (tsdf.ply)..." << std::endl;
	std::string name = "tsdf" + std::to_string(mnId) + ".ply";
  	SaveVoxelGrid2SurfacePointCloud(name, voxel_grid_dim_x, voxel_grid_dim_y, voxel_grid_dim_z, 
                                  voxel_size, voxel_grid_origin_x, voxel_grid_origin_y, voxel_grid_origin_z,
                                  voxel_grid_TSDF, voxel_grid_weight, 1.2f, 0.9f);

  	// Save TSDF voxel grid and its parameters to disk as binary file (float array)
  	//std::cout << "Saving TSDF voxel grid values to disk (tsdf.bin)..." << std::endl;
	name = "tsdf" + std::to_string(mnId) + ".bin";
  	std::string voxel_grid_saveto_path = name;
  	std::ofstream outFile(voxel_grid_saveto_path, std::ios::binary | std::ios::out);
  	float voxel_grid_dim_xf = (float) voxel_grid_dim_x;
  	float voxel_grid_dim_yf = (float) voxel_grid_dim_y;
  	float voxel_grid_dim_zf = (float) voxel_grid_dim_z;
  	outFile.write((char*)&voxel_grid_dim_xf, sizeof(float));
  	outFile.write((char*)&voxel_grid_dim_yf, sizeof(float));
  	outFile.write((char*)&voxel_grid_dim_zf, sizeof(float));
  	outFile.write((char*)&voxel_grid_origin_x, sizeof(float));
  	outFile.write((char*)&voxel_grid_origin_y, sizeof(float));
  	outFile.write((char*)&voxel_grid_origin_z, sizeof(float));
  	outFile.write((char*)&voxel_size, sizeof(float));
  	outFile.write((char*)&trunc_margin, sizeof(float));
  	for (int i = 0; i < voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z; ++i)
    	outFile.write((char*)&voxel_grid_TSDF[i], sizeof(float));
  	outFile.close();
}

void TSDF::Integrate(float *depth_im, std::vector<float> cam2world_vec) 
{

    // Read base frame camera pose
    std::copy(cam2world_vec.begin(), cam2world_vec.end(), cam2world);

    // Compute relative camera pose (camera-to-base frame)
    multiply_matrix(base2world_inv, cam2world, cam2base);
/*
	std::cout<<std::endl;
	std::cout<<cam_K[0*3+0]<<" "<<0           <<" "<<cam_K[0*3+2]<<std::endl;
	std::cout<<0           <<" "<<cam_K[1*3+1]<<" "<<cam_K[1*3+2]<<std::endl;
	std::cout<<std::endl;

	std::cout<<cam2world[0*4+0]<<" "<<cam2world[0*4+1]<<" "<< cam2world[0*4+2]<<" "<<cam2world[0*4+3]<<std::endl;
	std::cout<<cam2world[1*4+0]<<" "<<cam2world[1*4+1]<<" "<< cam2world[1*4+2]<<" "<<cam2world[1*4+3]<<std::endl;
	std::cout<<cam2world[2*4+0]<<" "<<cam2world[2*4+1]<<" "<< cam2world[2*4+2]<<" "<<cam2world[2*4+3]<<std::endl;
	std::cout<<cam2world[3*4+0]<<" "<<cam2world[3*4+1]<<" "<< cam2world[3*4+2]<<" "<<cam2world[3*4+3]<<std::endl;
	std::cout<<std::endl;

	std::cout<<cam2base[0*4+0]<<" "<<cam2base[0*4+1]<<" "<< cam2base[0*4+2]<<" "<<cam2base[0*4+3]<<std::endl;
	std::cout<<cam2base[1*4+0]<<" "<<cam2base[1*4+1]<<" "<< cam2base[1*4+2]<<" "<<cam2base[1*4+3]<<std::endl;
	std::cout<<cam2base[2*4+0]<<" "<<cam2base[2*4+1]<<" "<< cam2base[2*4+2]<<" "<<cam2base[2*4+3]<<std::endl;
	std::cout<<cam2base[3*4+0]<<" "<<cam2base[3*4+1]<<" "<< cam2base[3*4+2]<<" "<<cam2base[3*4+3]<<std::endl;
	std::cout<<std::endl;
*/
	hipMemcpy(gpu_cam2base, cam2base, 4 * 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpu_depth_im, depth_im, im_height * im_width * sizeof(float), hipMemcpyHostToDevice);
    checkCUDA(__LINE__, hipGetLastError());

    GpuIntegrate <<< voxel_grid_dim_z, voxel_grid_dim_y >>>(gpu_cam_K, gpu_cam2base, gpu_depth_im, im_height,
		im_width,  voxel_grid_dim_x, voxel_grid_dim_y, voxel_grid_dim_z, voxel_grid_origin_x, voxel_grid_origin_y,
		voxel_grid_origin_z, voxel_size, trunc_margin, gpu_voxel_grid_TSDF, gpu_voxel_grid_weight);
}

void TSDF::SaveVoxelGrid2SurfacePointCloud( const std::string &file_name, 
		int voxel_grid_dim_x, int voxel_grid_dim_y, int voxel_grid_dim_z,      float voxel_size, 
		float voxel_grid_origin_x, float voxel_grid_origin_y, float voxel_grid_origin_z, 
		float * voxel_grid_TSDF, float * voxel_grid_weight, float tsdf_thresh, float weight_thresh) 
{

	// Count total number of points in point cloud
	int num_pts = 0;
	for(int i = 0; i < voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z; i++)
	if(std::abs(voxel_grid_TSDF[i]) != 0.0f && voxel_grid_weight[i] > weight_thresh)
		num_pts++;

	//std::cout << "*********** Total number of points = " << num_pts++ << std::endl;

	// Create header for .ply file
	FILE *fp = fopen(file_name.c_str(), "w");
	fprintf(fp, "ply\n");
	fprintf(fp, "format binary_little_endian 1.0\n");
	fprintf(fp, "element vertex %d\n", num_pts);
	fprintf(fp, "property float x\n");
	fprintf(fp, "property float y\n");
	fprintf(fp, "property float z\n");
	fprintf(fp, "end_header\n");

	// Create point cloud content for ply file
	for(int i = 0; i < voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z; i++) 
	{
		// If TSDF value of voxel is less than some threshold, add voxel coordinates to point cloud
		if(std::abs(voxel_grid_TSDF[i]) != 0.0f && voxel_grid_weight[i] > weight_thresh) 
		{
			// Compute voxel indices in int for higher positive number range
			int z = floor(i / (voxel_grid_dim_x * voxel_grid_dim_y));
			int y = floor((i - (z * voxel_grid_dim_x * voxel_grid_dim_y)) / voxel_grid_dim_x);
			int x = i - (z * voxel_grid_dim_x * voxel_grid_dim_y) - (y * voxel_grid_dim_x);

			// Convert voxel indices to float, and save coordinates to ply file
			float pt_base_x = voxel_grid_origin_x + (float) x * voxel_size;
			float pt_base_y = voxel_grid_origin_y + (float) y * voxel_size;
			float pt_base_z = voxel_grid_origin_z + (float) z * voxel_size;

			fwrite(&pt_base_x, sizeof(float), 1, fp);
			fwrite(&pt_base_y, sizeof(float), 1, fp);
			fwrite(&pt_base_z, sizeof(float), 1, fp);
			
			//std::cout << pt_base_x << " " << pt_base_y << " " << pt_base_z << std::endl;
		}
  }
  fclose(fp);
}

// Load an M x N matrix from a text file (numbers delimited by spaces/tabs)
// Return the matrix as a float vector of the matrix in row-major order
std::vector<float> TSDF::LoadMatrixFromFile(std::string filename, int M, int N) 
{
  std::vector<float> matrix;
  FILE *fp = fopen(filename.c_str(), "r");
  for (int i = 0; i < M * N; i++) {
    float tmp;
    int iret = fscanf(fp, "%f", &tmp);
    matrix.push_back(tmp);
  }
  fclose(fp);
  return matrix;
}

// Read a depth image with size H x W and save the depth values (in meters) into a float array (in row-major order)
// The depth image file is assumed to be in 16-bit PNG format, depth in millimeters
void TSDF::ReadDepth(std::string filename, int H, int W, float * depth) 
{
  cv::Mat depth_mat = cv::imread(filename, CV_LOAD_IMAGE_UNCHANGED);
  if (depth_mat.empty()) {
    std::cout << "Error: depth image file not read!" << std::endl;
    cv::waitKey(0);
  }
  for (int r = 0; r < H; ++r)
    for (int c = 0; c < W; ++c) {
      depth[r * W + c] = (float)(depth_mat.at<unsigned short>(r, c)) / 1000.0f;
      if (depth[r * W + c] > 6.0f) // Only consider depth < 6m
        depth[r * W + c] = 0;
    }
}

// 4x4 matrix multiplication (matrices are stored as float arrays in row-major order)
void TSDF::multiply_matrix(const float m1[16], const float m2[16], float mOut[16]) {
  mOut[0]  = m1[0] * m2[0]  + m1[1] * m2[4]  + m1[2] * m2[8]   + m1[3] * m2[12];
  mOut[1]  = m1[0] * m2[1]  + m1[1] * m2[5]  + m1[2] * m2[9]   + m1[3] * m2[13];
  mOut[2]  = m1[0] * m2[2]  + m1[1] * m2[6]  + m1[2] * m2[10]  + m1[3] * m2[14];
  mOut[3]  = m1[0] * m2[3]  + m1[1] * m2[7]  + m1[2] * m2[11]  + m1[3] * m2[15];

  mOut[4]  = m1[4] * m2[0]  + m1[5] * m2[4]  + m1[6] * m2[8]   + m1[7] * m2[12];
  mOut[5]  = m1[4] * m2[1]  + m1[5] * m2[5]  + m1[6] * m2[9]   + m1[7] * m2[13];
  mOut[6]  = m1[4] * m2[2]  + m1[5] * m2[6]  + m1[6] * m2[10]  + m1[7] * m2[14];
  mOut[7]  = m1[4] * m2[3]  + m1[5] * m2[7]  + m1[6] * m2[11]  + m1[7] * m2[15];

  mOut[8]  = m1[8] * m2[0]  + m1[9] * m2[4]  + m1[10] * m2[8]  + m1[11] * m2[12];
  mOut[9]  = m1[8] * m2[1]  + m1[9] * m2[5]  + m1[10] * m2[9]  + m1[11] * m2[13];
  mOut[10] = m1[8] * m2[2]  + m1[9] * m2[6]  + m1[10] * m2[10] + m1[11] * m2[14];
  mOut[11] = m1[8] * m2[3]  + m1[9] * m2[7]  + m1[10] * m2[11] + m1[11] * m2[15];

  mOut[12] = m1[12] * m2[0] + m1[13] * m2[4] + m1[14] * m2[8]  + m1[15] * m2[12];
  mOut[13] = m1[12] * m2[1] + m1[13] * m2[5] + m1[14] * m2[9]  + m1[15] * m2[13];
  mOut[14] = m1[12] * m2[2] + m1[13] * m2[6] + m1[14] * m2[10] + m1[15] * m2[14];
  mOut[15] = m1[12] * m2[3] + m1[13] * m2[7] + m1[14] * m2[11] + m1[15] * m2[15];
}

// 4x4 matrix inversion (matrices are stored as float arrays in row-major order)
bool TSDF::invert_matrix(const float m[16], float invOut[16]) 
{
  float inv[16], det;
  int i;
  inv[0] = m[5]  * m[10] * m[15] -
           m[5]  * m[11] * m[14] -
           m[9]  * m[6]  * m[15] +
           m[9]  * m[7]  * m[14] +
           m[13] * m[6]  * m[11] -
           m[13] * m[7]  * m[10];

  inv[4] = -m[4]  * m[10] * m[15] +
           m[4]  * m[11] * m[14] +
           m[8]  * m[6]  * m[15] -
           m[8]  * m[7]  * m[14] -
           m[12] * m[6]  * m[11] +
           m[12] * m[7]  * m[10];

  inv[8] = m[4]  * m[9] * m[15] -
           m[4]  * m[11] * m[13] -
           m[8]  * m[5] * m[15] +
           m[8]  * m[7] * m[13] +
           m[12] * m[5] * m[11] -
           m[12] * m[7] * m[9];

  inv[12] = -m[4]  * m[9] * m[14] +
            m[4]  * m[10] * m[13] +
            m[8]  * m[5] * m[14] -
            m[8]  * m[6] * m[13] -
            m[12] * m[5] * m[10] +
            m[12] * m[6] * m[9];

  inv[1] = -m[1]  * m[10] * m[15] +
           m[1]  * m[11] * m[14] +
           m[9]  * m[2] * m[15] -
           m[9]  * m[3] * m[14] -
           m[13] * m[2] * m[11] +
           m[13] * m[3] * m[10];

  inv[5] = m[0]  * m[10] * m[15] -
           m[0]  * m[11] * m[14] -
           m[8]  * m[2] * m[15] +
           m[8]  * m[3] * m[14] +
           m[12] * m[2] * m[11] -
           m[12] * m[3] * m[10];

  inv[9] = -m[0]  * m[9] * m[15] +
           m[0]  * m[11] * m[13] +
           m[8]  * m[1] * m[15] -
           m[8]  * m[3] * m[13] -
           m[12] * m[1] * m[11] +
           m[12] * m[3] * m[9];

  inv[13] = m[0]  * m[9] * m[14] -
            m[0]  * m[10] * m[13] -
            m[8]  * m[1] * m[14] +
            m[8]  * m[2] * m[13] +
            m[12] * m[1] * m[10] -
            m[12] * m[2] * m[9];

  inv[2] = m[1]  * m[6] * m[15] -
           m[1]  * m[7] * m[14] -
           m[5]  * m[2] * m[15] +
           m[5]  * m[3] * m[14] +
           m[13] * m[2] * m[7] -
           m[13] * m[3] * m[6];

  inv[6] = -m[0]  * m[6] * m[15] +
           m[0]  * m[7] * m[14] +
           m[4]  * m[2] * m[15] -
           m[4]  * m[3] * m[14] -
           m[12] * m[2] * m[7] +
           m[12] * m[3] * m[6];

  inv[10] = m[0]  * m[5] * m[15] -
            m[0]  * m[7] * m[13] -
            m[4]  * m[1] * m[15] +
            m[4]  * m[3] * m[13] +
            m[12] * m[1] * m[7] -
            m[12] * m[3] * m[5];

  inv[14] = -m[0]  * m[5] * m[14] +
            m[0]  * m[6] * m[13] +
            m[4]  * m[1] * m[14] -
            m[4]  * m[2] * m[13] -
            m[12] * m[1] * m[6] +
            m[12] * m[2] * m[5];

  inv[3] = -m[1] * m[6] * m[11] +
           m[1] * m[7] * m[10] +
           m[5] * m[2] * m[11] -
           m[5] * m[3] * m[10] -
           m[9] * m[2] * m[7] +
           m[9] * m[3] * m[6];

  inv[7] = m[0] * m[6] * m[11] -
           m[0] * m[7] * m[10] -
           m[4] * m[2] * m[11] +
           m[4] * m[3] * m[10] +
           m[8] * m[2] * m[7] -
           m[8] * m[3] * m[6];

  inv[11] = -m[0] * m[5] * m[11] +
            m[0] * m[7] * m[9] +
            m[4] * m[1] * m[11] -
            m[4] * m[3] * m[9] -
            m[8] * m[1] * m[7] +
            m[8] * m[3] * m[5];

  inv[15] = m[0] * m[5] * m[10] -
            m[0] * m[6] * m[9] -
            m[4] * m[1] * m[10] +
            m[4] * m[2] * m[9] +
            m[8] * m[1] * m[6] -
            m[8] * m[2] * m[5];

  det = m[0] * inv[0] + m[1] * inv[4] + m[2] * inv[8] + m[3] * inv[12];

  if (det == 0)
    return false;

  det = 1.0 / det;

  for (i = 0; i < 16; i++)
    invOut[i] = inv[i] * det;

  return true;
}

void TSDF::FatalError(const int lineNumber) 
{
  std::cerr << "FatalError";
  if (lineNumber != 0) std::cerr << " at LINE " << lineNumber;
  std::cerr << ". Program Terminated." << std::endl;
  hipDeviceReset();
  exit(EXIT_FAILURE);
}

void TSDF::checkCUDA(const int lineNumber, hipError_t status) 
{
  if (status != hipSuccess) {
    std::cerr << "CUDA failure at LINE " << lineNumber << ": " << status << std::endl;
    FatalError();
  }
}

